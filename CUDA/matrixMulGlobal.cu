
#include <hip/hip_runtime.h>
//#include <algorithm>
//#include <cassert>
//#include <cstdlib>
//#include <functional>
//#include <iostream>
//#include <vector>
//#include <cuda_runtime.h>
//#include "device_launch_parameters.h"
//
//using std::cout;
//using std::generate;
//using std::vector;
//
//using namespace std;
//
//#define BLOCK_SIZE 32
//
////GPU side
////Each cell in output matrix has it's own thread to calculate it
//__global__ void matrixMul(int* a, int* b, int* c, int N) {
//	
//	int bx = blockIdx.x; // block number by x
//	int by = blockIdx.y; // block number by y
//	int tx = threadIdx.x; // thread number in block by x
//	int ty = threadIdx.y; // thread number in block by y
//	
//	int row = N * (BLOCK_SIZE * by + ty); //row from a
//	int col = BLOCK_SIZE * bx + tx; //col from b
//	int ic = row + col; //element num in c
//	int sum = 0;
//
//	//Calculate current row and column into corresponding cell of matrix c
//	for (int k = 0; k < N; k++) {
//		// Accumulate results for a single element
//		sum += a[row + k] * b[k * N + col];
//	}
//	c[ic] = sum;
//}
//
////CPU side
//int main() {
//	//Matrix size N x N
//	int N = 2048;
//
//	//Timer stuff
//	float time;
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start); 
//	cudaEventCreate(&stop);
//
//	//Matrix size in bytes
//	size_t byteSize = N * N * sizeof(int);
//
//	//Matrices
//	vector<int> h_a(N * N);
//	vector<int> h_b(N * N);
//	vector<int> h_c(N * N);
//
//	//Initialize matrices
//	generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
//	generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });
//
//	//Allocate device memory (device = GPU)
//	int* d_a, * d_b, * d_c;
//	cudaMalloc(&d_a, byteSize);
//	cudaMalloc(&d_b, byteSize);
//	cudaMalloc(&d_c, byteSize);
//
//	//Copy data to device
//	cudaMemcpy(d_a, h_a.data(), byteSize, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_b, h_b.data(), byteSize, cudaMemcpyHostToDevice);
//
//	//Blocks per grid dimension
//	int BlkGrdDim = (int)ceil(N / BLOCK_SIZE);
//
//	//dim3 - cuda int vector https://codeyarns.com/tech/2011-02-16-cuda-dim3.html
//	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
//	dim3 blocks(BlkGrdDim, BlkGrdDim);
//
//	//Start timer
//	cudaEventRecord(start, 0);
//
//	//Run kernel
//	matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);
//
//	cudaThreadSynchronize();
//	cudaEventRecord(stop, 0);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&time, start, stop);
//	cout << time << endl;
//
//	//Copy back to host
//	cudaMemcpy(h_c.data(), d_c, byteSize, cudaMemcpyDeviceToHost);
//
//	//Free memory on device
//	cudaFree(d_a);
//	cudaFree(d_b);
//	cudaFree(d_c);
//
//	//Event variables destruction (lol)
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//
//	cout << "Done" << endl;
//	
//	return 0;
//}