
#include <hip/hip_runtime.h>
//#include <algorithm>
//#include <cassert>
//#include <cstdlib>
//#include <functional>
//#include <iostream>
//#include <vector>
//#include <cuda_runtime.h>
//#include "device_launch_parameters.h"
//#include <random>
//
//using std::cout;
//using std::generate;
//using std::vector;
//
//using namespace std;
//
//#define BLOCK_SIZE 32
//
//__global__ void matrixMulKernel(float* a, float* b, float* c, int N) {
//	int gx = blockIdx.x * BLOCK_SIZE + threadIdx.x; // global thread x
//	int gy = blockIdx.y * BLOCK_SIZE + threadIdx.y; // global thread y
//
//	float sum = 0.f;
//
//	for (int r = 0; r < N; r++)
//	{
//		sum += a[gy * N + r] * b[gx + r * N];
//	}
//
//	c[gy * N + gx] = sum;
//}
//
//void printMatrix(vector<float> a, int N) {
//	for (int i = 0; i < N; i++) {
//		for (int j = 0; j < N; j++) {
//			cout << a[i * N + j] << " ";
//		}
//		cout << endl;
//	}
//	cout << endl;
//}
//
//vector<float> runMatrixMul(int N, vector<float> h_a, vector<float> h_b) {
//
//	// Matrix size in bytes
//	size_t byteSize = N * N * sizeof(float);
//
//	vector<float> h_c(N * N);
//
//	//Allocate device memory (device = GPU)
//	float* d_a, * d_b, * d_c;
//	cudaMalloc(&d_a, byteSize);
//	cudaMalloc(&d_b, byteSize);
//	cudaMalloc(&d_c, byteSize);
//
//	//Copy data to device
//	cudaMemcpy(d_a, h_a.data(), byteSize, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_b, h_b.data(), byteSize, cudaMemcpyHostToDevice);
//
//	//Blocks per grid dimension
//	int BlkGrdDim = (int)ceil((float)N / BLOCK_SIZE);
//
//	//dim3 - cuda int vector https://codeyarns.com/tech/2011-02-16-cuda-dim3.html
//	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
//	dim3 blocks(BlkGrdDim, BlkGrdDim);
//
//	//Run kernel
//	matrixMulKernel <<<blocks, threads>>> (d_a, d_b, d_c, N);
//	cudaThreadSynchronize();
//
//	//Copy back to host
//	cudaMemcpy(h_c.data(), d_c, byteSize, cudaMemcpyDeviceToHost);
//
//	//Free memory on device
//	cudaFree(d_a);
//	cudaFree(d_b);
//	cudaFree(d_c);
//
//	return h_c;
//}
//
////With timer for kernel analysis
//vector<float> runMatrixMul(int N, vector<float> h_a, vector<float> h_b, float* time) {
//
//	//Timer stuff
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//
//	// Matrix size in bytes
//	size_t byteSize = N * N * sizeof(float);
//
//	vector<float> h_c(N * N);
//
//	//Allocate device memory (device = GPU)
//	float* d_a, * d_b, * d_c;
//	cudaMalloc(&d_a, byteSize);
//	cudaMalloc(&d_b, byteSize);
//	cudaMalloc(&d_c, byteSize);
//
//	//Copy data to device
//	cudaMemcpy(d_a, h_a.data(), byteSize, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_b, h_b.data(), byteSize, cudaMemcpyHostToDevice);
//
//	//Blocks per grid dimension
//	int BlkGrdDim = (int)ceil((float)N / BLOCK_SIZE);
//
//	//dim3 - cuda int vector https://codeyarns.com/tech/2011-02-16-cuda-dim3.html
//	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
//	dim3 blocks(BlkGrdDim, BlkGrdDim);
//
//	//Start timer here
//	cudaEventRecord(start, 0);
//
//	//Run kernel
//	matrixMulKernel <<<blocks, threads>>> (d_a, d_b, d_c, N);
//	cudaThreadSynchronize();
//
//	//Stop timer here
//	cudaEventRecord(stop, 0);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(time, start, stop);
//
//	//Copy back to host
//	cudaMemcpy(h_c.data(), d_c, byteSize, cudaMemcpyDeviceToHost);
//
//	//Free memory on device
//	cudaFree(d_a);
//	cudaFree(d_b);
//	cudaFree(d_c);
//
//	return h_c;
//}
//
////CPU side
//int main() {
//	////Matrix size N x N
//	//const int N = 256;
//	//const int N = 512;
//	//const int N = 1024;
//	//const int N = 1536;
//	//const int N = 2048;
//	//const int N = 3072;
//	const int N = 4096;
//
//	vector<float> time_list;
//	int launchIter = 11;
//	int warmupLaunches = 1;
//
//	//Timer stuff
//	float time;
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//
//	//Matrices
//	vector<float> h_a(N * N);
//	vector<float> h_b(N * N);
//
//	for (int i = 0; i < launchIter; i++){
//		//Initialize matrices
//		generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
//		generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });
//
//		////Start timer here
//		//cudaEventRecord(start, 0);
//
//		//runMatrixMul(N, h_a, h_b);
//
//		////Stop timer here
//		//cudaEventRecord(stop, 0);
//		//cudaEventSynchronize(stop);
//		//cudaEventElapsedTime(&time, start, stop);
//
//		runMatrixMul(N, h_a, h_b, &time);
//
//		cout << "True time = " << time << endl;
//		time_list.push_back(time);
//	}
//
//	for (int i = 0; i < warmupLaunches; i++)
//		time_list.erase(time_list.begin());
//
//	float sumTime = 0;
//	for(auto el : time_list)
//	{
//		sumTime += el;
//	}
//
//	cout << endl << "Avg time = " << round(sumTime / (launchIter - warmupLaunches)) << endl;
//
//	//Event variables destruction (lol)
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//
//	cout << "Done" << endl;
//	return 0;
//}
