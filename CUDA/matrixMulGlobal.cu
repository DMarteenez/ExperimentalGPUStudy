#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include ""
#include <random>

using std::cout;
using std::generate;
using std::vector;

using namespace std;

#define BLOCK_SIZE 32


__global__ void matrixMul(float* a, float* b, float* c, int N) {
	int gx = blockIdx.x * BLOCK_SIZE + threadIdx.x; // global thread x
	int gy = blockIdx.y * BLOCK_SIZE + threadIdx.y; // global thread y

	float sum = 0.f;

	for (int r = 0; r < N; r++)
	{
		sum += a[gy * N + r] * b[gx + r * N];
	}

	c[gy * N + gx] = sum;
}

void printMatrix(vector<float> a, int N) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << a[i * N + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

//CPU side
int main() {
	////Matrix size N x N
	//const int N = 256;
	//const int N = 512;
	//const int N = 1024;
	//const int N = 1536;
	//const int N = 2048;
	//const int N = 3072;
	const int N = 4096;

	//Timer stuff
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Matrix size in bytes
	size_t byteSize = N * N * sizeof(float);

	//Matrices
	vector<float> h_a(N * N);
	vector<float> h_b(N * N);
	vector<float> h_c(N * N);

	//Initialize matrices
	generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
	generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

	//printMatrix(h_a, N);
	//printMatrix(h_b, N);

	//Start timer here
	hipEventRecord(start, 0);

	//Allocate device memory (device = GPU)
	float* d_a, * d_b, * d_c;
	hipMalloc(&d_a, byteSize);
	hipMalloc(&d_b, byteSize);
	hipMalloc(&d_c, byteSize);

	//Copy data to device
	hipMemcpy(d_a, h_a.data(), byteSize, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b.data(), byteSize, hipMemcpyHostToDevice);

	//Blocks per grid dimension
	int BlkGrdDim = (int)ceil((float)N / BLOCK_SIZE);

	//dim3 - cuda int vector https://codeyarns.com/tech/2011-02-16-cuda-dim3.html
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(BlkGrdDim, BlkGrdDim);

	//Run kernel
	matrixMul <<<blocks, threads>>> (d_a, d_b, d_c, N);
	hipDeviceSynchronize();

	//Copy back to host
	hipMemcpy(h_c.data(), d_c, byteSize, hipMemcpyDeviceToHost);

	//printMatrix(h_c, N);

	//Free memory on device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	//Stop timer here
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "Time = " << time << endl << endl;
	

	//Event variables destruction (lol)
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cout << "Done" << endl;

	return 0;
}
