#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include ""

using std::cout;
using std::generate;
using std::vector;

using namespace std;

#define BLOCK_SIZE 32

////GPU side
__global__ void matrixMul(float* a, float* b, float* c, int N) {
	int gy = threadIdx.y + blockIdx.y * BLOCK_SIZE;
	int gx = threadIdx.x + blockIdx.x * BLOCK_SIZE;
	int ly = threadIdx.y;
	int lx = threadIdx.x;
	float sum = 0.f;

	__shared__ float sA[BLOCK_SIZE][BLOCK_SIZE], sB[BLOCK_SIZE][BLOCK_SIZE];

	for (int k = 0; k < N; k += BLOCK_SIZE) {
		
		sA[ly][lx] = a[gy * N + lx + k];
		sB[ly][lx] = b[(ly + k) * N + gx];
		__syncthreads();
		for (int r = 0; r < BLOCK_SIZE; r++)
			sum += sA[ly][r] * sB[r][lx];
		__syncthreads();
	}
	c[gy * N + gx] = sum;
}

void printMatrix(vector<float> a, int N) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			cout << a[i * N + j] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

//CPU side
int main() {
	srand(time(0));
	//Matrix size N x N
	int N = 4096;

	//Timer stuff
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Matrix size in bytes
	size_t byteSize = N * N * sizeof(float);

	//Matrices
	vector<float> h_a(N * N);
	vector<float> h_b(N * N);
	vector<float> h_c(N * N);

	//Initialize matrices
	generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
	generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

	//printMatrix(h_a, N);
	//printMatrix(h_b, N);

	//Allocate device memory (device = GPU)
	float* d_a, * d_b, * d_c;
	hipMalloc(&d_a, byteSize);
	hipMalloc(&d_b, byteSize);
	hipMalloc(&d_c, byteSize);

	//Copy data to device
	hipMemcpy(d_a, h_a.data(), byteSize, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b.data(), byteSize, hipMemcpyHostToDevice);

	//Blocks per grid dimension
	int BlkGrdDim = (int)ceil((float)N / BLOCK_SIZE);

	//dim3 - cuda int vector https://codeyarns.com/tech/2011-02-16-cuda-dim3.html
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(BlkGrdDim, BlkGrdDim);

	//Start timer
	hipEventRecord(start, 0);

	//Run kernel
	matrixMul <<<blocks, threads>>> (d_a, d_b, d_c, N);

	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "Time = " << time << endl << endl;

	//Copy back to host
	hipMemcpy(h_c.data(), d_c, byteSize, hipMemcpyDeviceToHost);

	//printMatrix(h_c, N);

	//Free memory on device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	//Event variables destruction (lol)
	hipEventDestroy(start);
	hipEventDestroy(stop);

	cout << "Done" << endl;

	return 0;
}