
#include <hip/hip_runtime.h>
//#include <algorithm>
//#include <cassert>
//#include <cstdlib>
//#include <functional>
//#include <iostream>
//#include <vector>
//#include <cuda_runtime.h>
//#include "device_launch_parameters.h"
//
//using std::cout;
//using std::generate;
//using std::vector;
//
//using namespace std;
//
//#define BLOCK_SIZE 32
//
////GPU side
////My
//__global__ void matrixMul(float* a, float* b, float* c, int N) {
//	
//	int gx = blockIdx.x * BLOCK_SIZE + threadIdx.x; // global x in all threads of block
//	int gy = blockIdx.y * BLOCK_SIZE + threadIdx.y; // global y
//	int tx = threadIdx.x; // local x in thread
//	int ty = threadIdx.y; // local y
//	
//	float sum = 0.f;
//
//	for (int i = 0; i < N; i += BLOCK_SIZE)
//	{
//		__shared__ float as[BLOCK_SIZE][BLOCK_SIZE];
//		__shared__ float bs[BLOCK_SIZE][BLOCK_SIZE];
//		as[ty][tx] = a[gy * N + tx + i]; //copy from global mem to shared
//		bs[ty][tx] = b[(ty + i)*N + gx];
//		__syncthreads();
//		for (int k = 0; k < BLOCK_SIZE; k++) 
//			sum += as[ty][k] * bs[k][tx];
//		__syncthreads();
//	}
//	c[gy * N + gx] = sum;
//}
//
//////Not my
////__global__ void matrixMul(float* a, float* b, float* c, int N) {
////	int gi = threadIdx.y + blockIdx.y * BLOCK_SIZE;
////	int gj = threadIdx.x + blockIdx.x * BLOCK_SIZE;
////	int li = threadIdx.y;
////	int lj = threadIdx.x;
////	float sum = 0.f;
////
////	for (int k = 0; k < N; k += BLOCK_SIZE) {
////		__shared__ float sA[BLOCK_SIZE][BLOCK_SIZE], sB[BLOCK_SIZE][BLOCK_SIZE];
////		sA[li][lj] = a[gi * N + lj + k];
////		sB[li][lj] = b[(li + k) * N + gj];
////		__syncthreads();
////		for (int r = 0; r < BLOCK_SIZE; r++)
////			sum += sA[li][r] * sB[r][lj];
////		__syncthreads();
////	}
////	c[gi * N + gj] = sum;
////}
//
////CPU side
//int main() {
//	//Matrix size N x N
//	int N = 2048;
//
//	//Timer stuff
//	float time;
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start); 
//	cudaEventCreate(&stop);
//
//	//Matrix size in bytes
//	size_t byteSize = N * N * sizeof(float);
//
//	//Matrices
//	vector<float> h_a(N * N);
//	vector<float> h_b(N * N);
//	vector<float> h_c(N * N);
//
//	//Initialize matrices
//	generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
//	generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });
//
//	//Allocate device memory (device = GPU)
//	float* d_a, * d_b, * d_c;
//	cudaMalloc(&d_a, byteSize);
//	cudaMalloc(&d_b, byteSize);
//	cudaMalloc(&d_c, byteSize);
//
//	//Copy data to device
//	cudaMemcpy(d_a, h_a.data(), byteSize, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_b, h_b.data(), byteSize, cudaMemcpyHostToDevice);
//
//	//Blocks per grid dimension
//	int BlkGrdDim = (int)ceil(N / BLOCK_SIZE);
//
//	//dim3 - cuda int vector https://codeyarns.com/tech/2011-02-16-cuda-dim3.html
//	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
//	dim3 blocks(BlkGrdDim, BlkGrdDim);
//
//	//Start timer
//	cudaEventRecord(start, 0);
//
//	//Run kernel
//	matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);
//
//	cudaThreadSynchronize();
//	cudaEventRecord(stop, 0);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&time, start, stop);
//	cout << time << endl;
//
//	//Copy back to host
//	cudaMemcpy(h_c.data(), d_c, byteSize, cudaMemcpyDeviceToHost);
//
//	//Free memory on device
//	cudaFree(d_a);
//	cudaFree(d_b);
//	cudaFree(d_c);
//
//	//Event variables destruction (lol)
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//
//	cout << "Done" << endl;
//	
//	return 0;
//}